
#include <hip/hip_runtime.h>
#ifdef USE_CUDA

#define _USE_MATH_DEFINES
#include <cmath>

#include <Eigen/Dense>

#include <engine/Hamiltonian_Anisotropic.hpp>
#include <engine/Vectormath.hpp>
#include <data/Spin_System.hpp>

using std::vector;
using std::function;

using namespace Data;

namespace Engine
{
	Hamiltonian_Anisotropic::Hamiltonian_Anisotropic(
			scalarfield mu_s,
			intfield external_field_index, scalarfield external_field_magnitude, vectorfield external_field_normal,
			intfield anisotropy_index, scalarfield anisotropy_magnitude, vectorfield anisotropy_normal,
			std::vector<indexPairs> Exchange_indices, std::vector<scalarfield> Exchange_magnitude,
			std::vector<indexPairs> DMI_indices, std::vector<scalarfield> DMI_magnitude, std::vector<vectorfield> DMI_normal,
			std::vector<indexPairs> DD_indices, std::vector<scalarfield> DD_magnitude, std::vector<vectorfield> DD_normal,
			std::vector<indexQuadruplets> quadruplet_indices, std::vector<scalarfield> quadruplet_magnitude,
			std::vector<bool> boundary_conditions
	) :
		Hamiltonian(boundary_conditions),
		mu_s(mu_s),
		external_field_index(external_field_index), external_field_magnitude(external_field_magnitude), external_field_normal(external_field_normal),
		anisotropy_index(anisotropy_index), anisotropy_magnitude(anisotropy_magnitude), anisotropy_normal(anisotropy_normal),
		Exchange_indices(Exchange_indices), Exchange_magnitude(Exchange_magnitude),
		DMI_indices(DMI_indices), DMI_magnitude(DMI_magnitude), DMI_normal(DMI_normal),
		DD_indices(DD_indices), DD_magnitude(DD_magnitude), DD_normal(DD_normal),
		Quadruplet_indices(quadruplet_indices), Quadruplet_magnitude(quadruplet_magnitude)
	{
		// Renormalize the external field from Tesla to whatever
		for (unsigned int i = 0; i < external_field_magnitude.size(); ++i)
		{
			this->external_field_magnitude[i] = this->external_field_magnitude[i] * Vectormath::MuB() * mu_s[i];
		}

		this->Update_Energy_Contributions();
	}

	void Hamiltonian_Anisotropic::Update_Energy_Contributions()
	{
		this->E = std::vector<std::pair<std::string, scalar>>(0);
		// External field
		if (this->external_field_index.size() > 0)
		{
			this->E.push_back({"Zeeman", 0});
			this->idx_zeeman = this->E.size()-1;
		}
		else this->idx_zeeman = -1;
		// Anisotropy
		if (this->anisotropy_index.size() > 0)
		{
			this->E.push_back({"Anisotropy", 0});
			this->idx_anisotropy = this->E.size()-1;
		}
		else this->idx_anisotropy = -1;
		// Exchange
		if (this->Exchange_indices[0].size() > 0)
		{
			this->E.push_back({"Exchange", 0});
			this->idx_exchange = this->E.size()-1;
		}
		else this->idx_exchange = -1;
		// DMI
		if (this->DMI_indices[0].size() > 0)
		{
			this->E.push_back({"DMI", 0});
			this->idx_dmi = this->E.size()-1;
		}
		else this->idx_dmi = -1;
		// Dipole-Dipole
		if (this->DD_indices[0].size() > 0)
		{
			this->E.push_back({"DD", 0});
			this->idx_dd = this->E.size()-1;
		}
		else this->idx_dd = -1;
		// Quadruplet
		if (this->Quadruplet_indices[0].size() > 0)
		{
			this->E.push_back({"Quadruplet", 0});
			this->idx_quadruplet = this->E.size()-1;
		}
		else this->idx_quadruplet = -1;
	}

	scalar Hamiltonian_Anisotropic::Energy(const vectorfield & spins)
	{
		scalar sum = 0;
		auto e = Energy_Array(spins);
		for (auto E : e) sum += E.second;
		return sum;
	}

	std::vector<std::pair<std::string, scalar>> Hamiltonian_Anisotropic::Energy_Contributions(const vectorfield & spins)
	{
		// Set to zero
		for (auto& pair : this->E) pair.second = 0;

		// External field
		int nfields=this->anisotropy_index.size();
		cu_E_Zeeman<<<(nfields+255)/256,256>>>(spins, nfields, this->external_field_index.data(), this->external_field_magnitude.data(), this->external_field_normal.data(), E[idx_zeeman].second);
		if (this->idx_zeeman >=0 ) E_Zeeman(spins, E[idx_zeeman].second);

		// Anisotropy
		if (this->idx_anisotropy >=0 ) E_Anisotropy(spins, E[idx_anisotropy].second);

		// Pairs
		//		Loop over periodicity
		for (int i_periodicity = 0; i_periodicity < 8; ++i_periodicity)
		{
			// Check if boundary conditions contain this periodicity
			if ((i_periodicity == 0)
				|| (i_periodicity == 1 && this->boundary_conditions[0])
				|| (i_periodicity == 2 && this->boundary_conditions[1])
				|| (i_periodicity == 3 && this->boundary_conditions[2])
				|| (i_periodicity == 4 && this->boundary_conditions[0] && this->boundary_conditions[1])
				|| (i_periodicity == 5 && this->boundary_conditions[0] && this->boundary_conditions[2])
				|| (i_periodicity == 6 && this->boundary_conditions[1] && this->boundary_conditions[2])
				|| (i_periodicity == 7 && this->boundary_conditions[0] && this->boundary_conditions[1] && this->boundary_conditions[2]))
			{
				//		Energies of this periodicity
				// Exchange
				if (this->idx_exchange >=0 ) E_Exchange(spins, Exchange_indices[i_periodicity], Exchange_magnitude[i_periodicity], E[idx_exchange].second);
				// DMI
				if (this->idx_dmi >=0 ) E_DMI(spins, DMI_indices[i_periodicity], DMI_magnitude[i_periodicity], DMI_normal[i_periodicity], E[idx_dmi].second);
				// DD
				if (this->idx_dd >=0 ) E_DD(spins, DD_indices[i_periodicity], DD_magnitude[i_periodicity], DD_normal[i_periodicity], E[idx_dd].second);
				// Quadruplet
				if (this->idx_quadruplet >=0 ) E_Quadruplet(spins, Quadruplet_indices[i_periodicity], Quadruplet_magnitude[i_periodicity], E[idx_quadruplet].second);
			}
		}

		// Return
		return this->E;
	}


	__global__ void cu_E_Zeeman(Vector3 *spins, int nfields, int *external_field_index, scalar *external_field_magnitude, Vector3 *external_field_normal, scalar *Energy)
	{
		for (int ifield = blockIdx.x * blockDim.x + threadIdx.x; ifield < nfields; ifield += blockDim.x * gridDim.x) 
		{
			int ispin = external_field_index[ifield];
			atomicAdd(Energy[ispin], - external_field_magnitude[ifield] * external_field_normal[ifield].dot(spins[ispin]));
		}
	}



	__global__ void Hamiltonian_Anisotropic::cu_E_Zeeman(Vector3 *spins, int *external_field_index, scalar *external_field_magnitude, Vector3 *external_field_normal, scalar *E , size_t N)
	{
		int idx = blockIdx.x * blockDim.x + threadIdx.x;
		if(idx < N)
		{
			atomicAdd(E[external_field_index[idx]], - external_field_magnitude[idx] * external_field_normal[idx].dot(spins[external_field_index[idx]]));
		}
	}
	void Hamiltonian_Anisotropic::E_Zeeman(const vectorfield & spins, scalar & Energy)
	{
		for (unsigned int i = 0; i < this->external_field_index.size(); ++i)
		{
			cu_scale<<<(n+1023)/1024, 1024>>>(spins.data(), this->external_field_index.data(), this->external_field_magnitude.data(), this->external_field_normal.data(), Energy, n);
		}
	}

	void Hamiltonian_Anisotropic::E_Anisotropy(const vectorfield & spins, scalar & Energy)
	{
		for (unsigned int i = 0; i < this->anisotropy_index.size(); ++i)
		{
			Energy -= this->anisotropy_magnitude[i] * std::pow(anisotropy_normal[i].dot(spins[anisotropy_index[i]]), 2.0);
		}
	}

	void Hamiltonian_Anisotropic::E_Exchange(const vectorfield & spins, indexPairs & indices, scalarfield & J_ij, scalar & Energy)
	{
		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			Energy -= J_ij[i_pair] * spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]]);
		}
	}

	void Hamiltonian_Anisotropic::E_DMI(const vectorfield & spins, indexPairs & indices, scalarfield & DMI_magnitude, vectorfield & DMI_normal, scalar & Energy)
	{
		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			Energy -= DMI_magnitude[i_pair] * DMI_normal[i_pair].dot(spins[indices[i_pair][0]].cross(spins[indices[i_pair][1]]));
		}
	}

	void Hamiltonian_Anisotropic::E_DD(const vectorfield & spins, indexPairs & indices, scalarfield & DD_magnitude, vectorfield & DD_normal, scalar & Energy)
	{
		//scalar mult = -Utility::Vectormath::MuB()*Utility::Vectormath::MuB()*1.0 / 4.0 / M_PI; // multiply with mu_B^2
		scalar mult = 0.0536814951168; // mu_0*mu_B**2/(4pi*10**-30) -- the translations are in angstr�m, so the |r|[m] becomes |r|[m]*10^-10
		scalar result = 0.0;

		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			if (DD_magnitude[i_pair] > 0.0)
			{
				Energy -= mult / std::pow(DD_magnitude[i_pair], 3.0) *
					(3 * spins[indices[i_pair][1]].dot(DD_normal[i_pair]) * spins[indices[i_pair][0]].dot(DD_normal[i_pair]) - spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]]));
			}

		}
	}// end DipoleDipole


	void Hamiltonian_Anisotropic::E_Quadruplet(const vectorfield & spins, indexQuadruplets & indices, scalarfield & magnitude, scalar & Energy)
	{
		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			Energy -= magnitude[i_pair] * (spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]])) * (spins[indices[i_pair][2]].dot(spins[indices[i_pair][3]]));
		}
	}



	void Hamiltonian_Anisotropic::Gradient(const vectorfield & spins, vectorfield & gradient)
	{
		int nos = spins.size();
		// Loop over Spins
		for (int i = 0; i < nos; ++i)
		{
			gradient[i].setZero();
		}

		// External field
		Gradient_Zeeman(spins, gradient);

		// Anisotropy
		Gradient_Anisotropy(spins, gradient);

		// Pairs
		//		Loop over periodicity
		for (int i_periodicity = 0; i_periodicity < 8; ++i_periodicity)
		{
			// Check if boundary conditions contain this periodicity
			if ((i_periodicity == 0)
				|| (i_periodicity == 1 && this->boundary_conditions[0])
				|| (i_periodicity == 2 && this->boundary_conditions[1])
				|| (i_periodicity == 3 && this->boundary_conditions[2])
				|| (i_periodicity == 4 && this->boundary_conditions[0] && this->boundary_conditions[1])
				|| (i_periodicity == 5 && this->boundary_conditions[0] && this->boundary_conditions[2])
				|| (i_periodicity == 6 && this->boundary_conditions[1] && this->boundary_conditions[2])
				|| (i_periodicity == 7 && this->boundary_conditions[0] && this->boundary_conditions[1] && this->boundary_conditions[2]))
			{
				//		Gradients of this periodicity
				// Exchange
				this->Gradient_Exchange(spins, Exchange_indices[i_periodicity], Exchange_magnitude[i_periodicity], gradient);
				// DMI
				this->Gradient_DMI(spins, DMI_indices[i_periodicity], DMI_magnitude[i_periodicity], DMI_normal[i_periodicity], gradient);
				// DD
				this->Gradient_DD(spins, DD_indices[i_periodicity], DD_magnitude[i_periodicity], DD_normal[i_periodicity], gradient);
				// Quadruplet
				this->Gradient_Quadruplet(spins, Quadruplet_indices[i_periodicity], Quadruplet_magnitude[i_periodicity], gradient);
			}
		}

		// Triplet Interactions

		// Quadruplet Interactions
	}

	void Hamiltonian_Anisotropic::Gradient_Zeeman(const vectorfield & spins, vectorfield & gradient)
	{
		for (unsigned int i = 0; i < this->external_field_index.size(); ++i)
		{
			eff_field[external_field_index[i]] -= this->external_field_magnitude[i] * this->external_field_normal[i];
		}
	}

	void Hamiltonian_Anisotropic::Gradient_Anisotropy(const vectorfield & spins, vectorfield & gradient)
	{
		for (unsigned int i = 0; i < this->anisotropy_index.size(); ++i)
		{
			eff_field[anisotropy_index[i]] -= 2.0 * this->anisotropy_magnitude[i] * this->anisotropy_normal[i] * anisotropy_normal[i].dot(spins[anisotropy_index[i]]);
		}
	}

	void Hamiltonian_Anisotropic::Gradient_Exchange(const vectorfield & spins, indexPairs & indices, scalarfield & J_ij, vectorfield & gradient)
	{
		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			eff_field[indices[i_pair][0]] -= J_ij[i_pair] * spins[indices[i_pair][1]];
			eff_field[indices[i_pair][1]] -= J_ij[i_pair] * spins[indices[i_pair][0]];
		}
	}

	void Hamiltonian_Anisotropic::Gradient_DMI(const vectorfield & spins, indexPairs & indices, scalarfield & DMI_magnitude, vectorfield & DMI_normal, vectorfield & gradient)
	{
		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			eff_field[indices[i_pair][0]] -= DMI_magnitude[i_pair] * spins[indices[i_pair][1]].cross(DMI_normal[i_pair]);
			eff_field[indices[i_pair][1]] += DMI_magnitude[i_pair] * spins[indices[i_pair][0]].cross(DMI_normal[i_pair]);
		}
	}

	void Hamiltonian_Anisotropic::Gradient_DD(const vectorfield & spins, indexPairs & indices, scalarfield & DD_magnitude, vectorfield & DD_normal, vectorfield & gradient)
	{
		//scalar mult = Utility::Vectormath::MuB()*Utility::Vectormath::MuB()*1.0 / 4.0 / M_PI; // multiply with mu_B^2
		scalar mult = 0.0536814951168; // mu_0*mu_B**2/(4pi*10**-30) -- the translations are in angstr�m, so the |r|[m] becomes |r|[m]*10^-10
		
		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			if (DD_magnitude[i_pair] > 0.0)
			{
				scalar skalar_contrib = mult / std::pow(DD_magnitude[i_pair], 3.0);
				eff_field[indices[i_pair][0]] -= skalar_contrib * (3 * DD_normal[i_pair] * spins[indices[i_pair][1]].dot(DD_normal[i_pair]) - spins[indices[i_pair][1]]);
				eff_field[indices[i_pair][1]] -= skalar_contrib * (3 * DD_normal[i_pair] * spins[indices[i_pair][0]].dot(DD_normal[i_pair]) - spins[indices[i_pair][0]]);
			}
		}
	}//end Field_DipoleDipole


	void Hamiltonian_Anisotropic::Gradient_Quadruplet(const vectorfield & spins, indexQuadruplets & indices, scalarfield & magnitude, vectorfield & gradient)
	{
		for (unsigned int i_pair = 0; i_pair < indices.size(); ++i_pair)
		{
			eff_field[indices[i_pair][0]] -= magnitude[i_pair] * spins[indices[i_pair][1]] * (spins[indices[i_pair][2]].dot(spins[indices[i_pair][3]]));
			eff_field[indices[i_pair][1]] -= magnitude[i_pair] * spins[indices[i_pair][0]] *  (spins[indices[i_pair][2]].dot(spins[indices[i_pair][3]]));
			eff_field[indices[i_pair][2]] -= magnitude[i_pair] * (spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]])) * spins[indices[i_pair][3]];
			eff_field[indices[i_pair][3]] -= magnitude[i_pair] * (spins[indices[i_pair][0]].dot(spins[indices[i_pair][1]])) * spins[indices[i_pair][2]];
		}
	}


	void Hamiltonian_Anisotropic::Hessian(const vectorfield & spins, MatrixX & hessian)
	{
		int nos = spins.size();

		// Set to zero
		// for (auto& h : hessian) h = 0;
		hessian.setZero();

		// Single Spin elements
		for (int alpha = 0; alpha < 3; ++alpha)
		{
			for (unsigned int i = 0; i < anisotropy_index.size(); ++i)
			{
				int idx = anisotropy_index[i];
				// scalar x = -2.0*this->anisotropy_magnitude[i] * std::pow(this->anisotropy_normal[i][alpha], 2);
				hessian(3*idx + alpha, 3*idx + alpha) += -2.0*this->anisotropy_magnitude[i]*std::pow(this->anisotropy_normal[i][alpha],2);
			}
		}

		// std::cerr << "calculated hessian" << std::endl;

		// // Spin Pair elements
		// for (int i_periodicity = 0; i_periodicity < 8; ++i_periodicity)
		// {
		// 	//		Check if boundary conditions contain this periodicity
		// 	if ((i_periodicity == 0)
		// 		|| (i_periodicity == 1 && this->boundary_conditions[0])
		// 		|| (i_periodicity == 2 && this->boundary_conditions[1])
		// 		|| (i_periodicity == 3 && this->boundary_conditions[2])
		// 		|| (i_periodicity == 4 && this->boundary_conditions[0] && this->boundary_conditions[1])
		// 		|| (i_periodicity == 5 && this->boundary_conditions[0] && this->boundary_conditions[2])
		// 		|| (i_periodicity == 6 && this->boundary_conditions[1] && this->boundary_conditions[2])
		// 		|| (i_periodicity == 7 && this->boundary_conditions[0] && this->boundary_conditions[1] && this->boundary_conditions[2]))
		// 	{
		// 		//		Loop over pairs of this periodicity
		// 		// Exchange
		// 		for (unsigned int i_pair = 0; i_pair < this->Exchange_indices[i_periodicity].size(); ++i_pair)
		// 		{
		// 			for (int alpha = 0; alpha < 3; ++alpha)
		// 			{
		// 				int idx_i = 3*Exchange_indices[i_periodicity][i_pair][0] + alpha;
		// 				int idx_j = 3*Exchange_indices[i_periodicity][i_pair][1] + alpha;
		// 				hessian(idx_i,idx_j) += -Exchange_magnitude[i_periodicity][i_pair];
		// 				hessian(idx_j,idx_i) += -Exchange_magnitude[i_periodicity][i_pair];
		// 			}
		// 		}
		// 		// DMI
		// 		for (unsigned int i_pair = 0; i_pair < this->DMI_indices[i_periodicity].size(); ++i_pair)
		// 		{
		// 			for (int alpha = 0; alpha < 3; ++alpha)
		// 			{
		// 				for (int beta = 0; beta < 3; ++beta)
		// 				{
		// 					int idx_i = 3*DMI_indices[i_periodicity][i_pair][0] + alpha;
		// 					int idx_j = 3*DMI_indices[i_periodicity][i_pair][1] + beta;
		// 					if ( (alpha == 0 && beta == 1) || (alpha == 1 && beta == 0) )
		// 					{
		// 						hessian(idx_i,idx_j) +=
		// 							DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][2];
		// 						hessian(idx_j,idx_i) +=
		// 							DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][2];
		// 					}
		// 					else if ( (alpha == 0 && beta == 2) || (alpha == 2 && beta == 0) )
		// 					{
		// 						hessian(idx_i,idx_j) +=
		// 							-DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][1];
		// 						hessian(idx_j,idx_i) +=
		// 							-DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][1];
		// 					}
		// 					else if ( (alpha == 1 && beta == 2) || (alpha == 2 && beta == 1) )
		// 					{
		// 						hessian(idx_i,idx_j) +=
		// 							DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][0];
		// 						hessian(idx_j,idx_i) +=
		// 							DMI_magnitude[i_periodicity][i_pair] * DMI_normal[i_periodicity][i_pair][0];
		// 					}
		// 				}
		// 			}
		// 		}
		// //		// Dipole-Dipole
		// //		for (unsigned int i_pair = 0; i_pair < this->DD_indices[i_periodicity].size(); ++i_pair)
		// //		{
		// //			// indices
		// //			int idx_1 = DD_indices[i_periodicity][i_pair][0];
		// //			int idx_2 = DD_indices[i_periodicity][i_pair][1];
		// //			// prefactor
		// //			scalar prefactor = 0.0536814951168
		// //				* this->mu_s[idx_1] * this->mu_s[idx_2]
		// //				/ std::pow(DD_magnitude[i_periodicity][i_pair], 3);
		// //			// components
		// //			for (int alpha = 0; alpha < 3; ++alpha)
		// //			{
		// //				for (int beta = 0; beta < 3; ++beta)
		// //				{
		// //					int idx_h = idx_1 + alpha*nos + 3 * nos*(idx_2 + beta*nos);
		// //					if (alpha == beta)
		// //						hessian[idx_h] += prefactor;
		// //					hessian[idx_h] += -3.0*prefactor*DD_normal[i_periodicity][i_pair][alpha] * DD_normal[i_periodicity][i_pair][beta];
		// //				}
		// //			}
		// //		}
		// 	}// end if periodicity
		// }// end for periodicity
	}

	// Hamiltonian name as string
	static const std::string name = "Anisotropic Heisenberg";
	const std::string& Hamiltonian_Anisotropic::Name() { return name; }
}

#endif